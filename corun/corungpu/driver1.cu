#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// helper functions and utilities to work with CUDA
#define ERT_TRIALS_MIN      1
#define ERT_WORKING_SET_MIN 1
#define GBUNIT              (1024 * 1024 * 1024)

// Usage:
// make clean && make CFLAGS="-DERT_FLOP=64 -DFP32" && ./main

#define REP2(S)                                                                                    \
    S;                                                                                             \
    S
#define REP4(S)                                                                                    \
    REP2(S);                                                                                       \
    REP2(S)
#define REP8(S)                                                                                    \
    REP4(S);                                                                                       \
    REP4(S)
#define REP16(S)                                                                                   \
    REP8(S);                                                                                       \
    REP8(S)
#define REP32(S)                                                                                   \
    REP16(S);                                                                                      \
    REP16(S)
#define REP64(S)                                                                                   \
    REP32(S);                                                                                      \
    REP32(S)
#define REP128(S)                                                                                  \
    REP64(S);                                                                                      \
    REP64(S)
#define REP256(S)                                                                                  \
    REP128(S);                                                                                     \
    REP128(S)
#define REP512(S)                                                                                  \
    REP256(S);                                                                                     \
    REP256(S)

#define KERNEL2(a, b, c) ((a) = (a) * (b) + (c))
#define KERNEL1(a, b, c) ((a) = (b) + (c))

#if FP64
void initialize(uint64_t nsize, double* __restrict__ A, double value)
#elif FP32
void initialize(uint64_t nsize, float* __restrict__ A, float value)
#elif FP16
void initialize(uint64_t nsize, __half* __restrict__ A, __half value)
#else
void initialize(uint64_t nsize, double* __restrict__ A, double value)
#endif
{
    uint64_t i;
    for (i = 0; i < nsize; ++i) {
        A[i] = value;
    }
}

#if FP64
void gpuKernel(uint64_t nsize, uint64_t ntrials, double* __restrict__ array, int* bytes_per_elem,
               int* mem_accesses_per_elem);
#elif FP32
void gpuKernel(uint64_t nsize, uint64_t ntrials, float* __restrict__ array, int* bytes_per_elem,
               int* mem_accesses_per_elem);
#elif FP16
void gpuKernel(uint64_t nsize, uint64_t ntrials, __half* __restrict__ array, int* bytes_per_elem,
               int* mem_accesses_per_elem);
#else
void gpuKernel(uint64_t nsize, uint64_t ntrials, double* __restrict__ array, int* bytes_per_elem,
               int* mem_accesses_per_elem);
#endif

#if FP64
__global__ void block_stride(uint64_t ntrials, uint64_t nsize, double* A)
#elif FP32
__global__ void block_stride(uint64_t ntrials, uint64_t nsize, float* A)
#elif FP16
__global__ void block_stride(uint64_t ntrials, uint64_t nsize, __half* A)
#else
__global__ void block_stride(uint64_t ntrials, uint64_t nsize, double* A)
#endif
{
    uint64_t total_thr = gridDim.x * blockDim.x;
    uint64_t elem_per_thr = (nsize + (total_thr - 1)) / total_thr;
    uint64_t blockOffset = blockIdx.x * blockDim.x;

    uint64_t start_idx = blockOffset + threadIdx.x;
    uint64_t end_idx = start_idx + elem_per_thr * total_thr;
    uint64_t stride_idx = total_thr;

    if (start_idx > nsize) {
        start_idx = nsize;
    }

    if (end_idx > nsize) {
        end_idx = nsize;
    }

#if FP64
    double alpha = 0.5;
#elif FP32
    float alpha = 0.5;
#elif FP16
    __half alpha = 0.5;
#else
    double alpha = 0.5;
#endif

    uint64_t i, j;
    for (j = 0; j < ntrials; ++j) {
        for (i = start_idx; i < end_idx; i += stride_idx) {
#if FP64
            double beta = 0.8;
#elif FP32
            float beta = 0.8;
#elif FP16
            __half beta = 0.8;
#else
            double beta = 0.8;
#endif

#if ((ERT_FLOP & 1) == 1) /* add 1 flop */
            KERNEL1(beta, A[i], alpha);
#endif
#if ((ERT_FLOP & 2) == 2) /* add 2 flops */
            KERNEL2(beta, A[i], alpha);
#endif
#if ((ERT_FLOP & 4) == 4) /* add 4 flops */
            REP2(KERNEL2(beta, A[i], alpha));
#endif
#if ((ERT_FLOP & 8) == 8) /* add 8 flops */
            REP4(KERNEL2(beta, A[i], alpha));
#endif
#if ((ERT_FLOP & 16) == 16) /* add 16 flops */
            REP8(KERNEL2(beta, A[i], alpha));
#endif
#if ((ERT_FLOP & 32) == 32) /* add 32 flops */
            REP16(KERNEL2(beta, A[i], alpha));
#endif
#if ((ERT_FLOP & 64) == 64) /* add 64 flops */
            REP32(KERNEL2(beta, A[i], alpha));
#endif
#if ((ERT_FLOP & 128) == 128) /* add 128 flops */
            REP64(KERNEL2(beta, A[i], alpha));
#endif
#if ((ERT_FLOP & 256) == 256) /* add 256 flops */
            REP128(KERNEL2(beta, A[i], alpha));
#endif
#if ((ERT_FLOP & 512) == 512) /* add 512 flops */
            REP256(KERNEL2(beta, A[i], alpha));
#endif
#if ((ERT_FLOP & 1024) == 1024) /* add 1024 flops */
            REP512(KERNEL2(beta, A[i], alpha));
#endif

            A[i] = beta;
        }
        alpha = alpha * (1 - 1e-8);
    }
}
int gpu_blocks = 512;
int gpu_threads = 512;

#if FP64
void gpuKernel(uint64_t nsize, uint64_t ntrials, double* __restrict__ A, int* bytes_per_elem,
               int* mem_accesses_per_elem)
#elif FP32
void gpuKernel(uint64_t nsize, uint64_t ntrials, float* __restrict__ A, int* bytes_per_elem,
               int* mem_accesses_per_elem)
#elif FP16
void gpuKernel(uint64_t nsize, uint64_t ntrials, __half* __restrict__ A, int* bytes_per_elem,
               int* mem_accesses_per_elem)
#else
void gpuKernel(uint64_t nsize, uint64_t ntrials, double* __restrict__ A, int* bytes_per_elem,
               int* mem_accesses_per_elem)
#endif
{
    *bytes_per_elem = sizeof(*A);
    *mem_accesses_per_elem = 2;
    // gpu_blocks = (nsize+1023)/1024;
    block_stride<<<gpu_blocks, gpu_threads>>>(ntrials, nsize, A);
}

double getTime() {
    double time;
    struct timeval tm;
    gettimeofday(&tm, NULL);
    time = tm.tv_sec + (tm.tv_usec / 1000000.0);
    return time;
}

int main(int argc, char* argv[]) {

    int rank = 0;
    int nprocs = 1;
    int nthreads = 1;
    int id = 0;

    uint64_t TSIZE = 1 << 30;
    uint64_t PSIZE = TSIZE / nprocs;

#if FP64
    double* buf = (double*)malloc(PSIZE);
#elif FP32
    float* buf = (float*)malloc(PSIZE);
#elif FP16
    __half* buf = (__half*)malloc(PSIZE);
#else
    double* buf = (double*)malloc(PSIZE);
#endif
    printf("nsize,trials,microseconds,bytes,single_thread_bandwidth,total_bandwidth,GFLOPS,"
           "bandwidth(GB/s)\n");

    if (buf == NULL) {
        fprintf(stderr, "Out of memory!\n");
        return -1;
    }

    {
        id = 0;
        nthreads = 1;

        int num_gpus = 0;
        int gpu;
        int gpu_id;
        int numSMs;

        hipGetDeviceCount(&num_gpus);
        if (num_gpus < 1) {
            fprintf(stderr, "No CUDA device detected.\n");
            return -1;
        }

        for (gpu = 0; gpu < num_gpus; gpu++) {
            hipDeviceProp_t dprop;
            hipGetDeviceProperties(&dprop, gpu);
            /* printf("%d: %s\n",gpu,dprop.name); */
        }

        hipSetDevice(id % num_gpus);
        hipGetDevice(&gpu_id);
        hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, gpu_id);

        uint64_t nsize = PSIZE / nthreads;
        nsize = nsize & (~(64 - 1));
#if FP64
        nsize = nsize / sizeof(double);
#elif FP32
        nsize = nsize / sizeof(float);
#elif FP16
        nsize = nsize / sizeof(__half);
#else
        nsize = nsize / sizeof(double);
#endif
        uint64_t nid = nsize * id;

        // initialize small chunck of buffer within each thread
        initialize(nsize, &buf[nid], 1.0);

#if FP64
        double* d_buf;
        hipMalloc((void**)&d_buf, nsize * sizeof(double));
        hipMemset(d_buf, 0, nsize * sizeof(double));
#elif FP32
        float* d_buf;
        hipMalloc((void**)&d_buf, nsize * sizeof(float));
        hipMemset(d_buf, 0, nsize * sizeof(float));
#elif FP16
        __half* d_buf;
        hipMalloc((void**)&d_buf, nsize * sizeof(__half));
        hipMemset(d_buf, 0, nsize * sizeof(__half));
#else
        double* d_buf;
        hipMalloc((void**)&d_buf, nsize * sizeof(double));
        hipMemset(d_buf, 0, nsize * sizeof(double));
#endif

        hipDeviceSynchronize();

        double startTime, endTime;
        uint64_t n, nNew;
        uint64_t t;
        int bytes_per_elem;
        int mem_accesses_per_elem;

        n = 1 << 22;
        while (n <= nsize) { // working set - nsize
            uint64_t ntrials = nsize / n;
            if (ntrials < 1)
                ntrials = 1;
            // 600 original
            for (t = 1; t <= 600; t = t + 1) { // working set - ntrials
#if FP64
                hipMemcpy(d_buf, &buf[nid], n * sizeof(double), hipMemcpyHostToDevice);
#elif FP32
                hipMemcpy(d_buf, &buf[nid], n * sizeof(float), hipMemcpyHostToDevice);
#elif FP16
                hipMemcpy(d_buf, &buf[nid], n * sizeof(__half), hipMemcpyHostToDevice);
#else
                hipMemcpy(d_buf, &buf[nid], n * sizeof(double), hipMemcpyHostToDevice);
#endif

                hipDeviceSynchronize();

                if ((id == 0) && (rank == 0)) {
                    startTime = getTime();
                }

                gpuKernel(n, t, d_buf, &bytes_per_elem, &mem_accesses_per_elem);
                hipDeviceSynchronize();

                if ((id == 0) && (rank == 0)) {
                    endTime = getTime();
                    double seconds = (double)(endTime - startTime);
                    uint64_t working_set_size = n * nthreads * nprocs;
                    uint64_t total_bytes =
                        t * working_set_size * bytes_per_elem * mem_accesses_per_elem;
                    uint64_t total_flops = t * working_set_size * ERT_FLOP;
                    // printf("thread: %d\n", nthreads);
                    // nsize; trials; microseconds; bytes; single thread bandwidth; total bandwidth
                    printf("%lu,%lu,%.3lf,%lu,%lu,%.3lf,%.3lf\n", working_set_size * bytes_per_elem,
                           t, seconds * 1000000, total_bytes, total_flops,
                           total_flops / seconds / 1e9,
                           total_bytes * 1.0 / seconds / 1024 / 1024 / 1024);
                } // print

#if FP64
                hipMemcpy(&buf[nid], d_buf, n * sizeof(double), hipMemcpyDeviceToHost);
#elif FP32
                hipMemcpy(&buf[nid], d_buf, n * sizeof(float), hipMemcpyDeviceToHost);
#elif FP16
                hipMemcpy(&buf[nid], d_buf, n * sizeof(__half), hipMemcpyDeviceToHost);
#else
                hipMemcpy(&buf[nid], d_buf, n * sizeof(double), hipMemcpyDeviceToHost);
#endif
                hipDeviceSynchronize();
            } // working set - ntrials

            nNew = 1.1 * n;
            if (nNew == n) {
                nNew = n + 1;
            }

            n = nNew;
            // no break brfore
            break;
        } // working set - nsize

        hipFree(d_buf);

        if (hipGetLastError() != hipSuccess) {
            printf("Last cuda error: %s\n", hipGetErrorString(hipGetLastError()));
        }

        hipDeviceReset();
    } // parallel region

    free(buf);

    printf("\n");
    printf("META_DATA\n");
    printf("FLOPS          %d\n", ERT_FLOP);

    printf("GPU_BLOCKS     %d\n", gpu_blocks);
    printf("GPU_THREADS    %d\n", gpu_threads);

    return 0;
}
